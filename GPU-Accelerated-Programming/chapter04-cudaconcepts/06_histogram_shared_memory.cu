#include <stdio.h>
#include <hip/hip_runtime.h>

#define SIZE 1000
#define NUM_BIN 256

__global__ void histogram_shared_memory(int *d_b, int *d_a)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    int offset = blockDim.x * gridDim.x;
    __shared__ int cache[256];
    cache[threadIdx.x] = 0;
    __syncthreads();
    
    while (tid < SIZE)
    {
        atomicAdd(&(cache[d_a[tid]]), 1);
        tid += offset;
    }
    __syncthreads();
    atomicAdd(&(d_b[threadIdx.x]), cache[threadIdx.x]);
}

int main()
{
    // generate the input array on the host
    int h_a[SIZE];
    for (int i = 0; i < SIZE; i++) {
        //h_a[i] = bit_reverse(i, log2(SIZE));
        h_a[i] = i % NUM_BIN;
    }
    int h_b[NUM_BIN];
    for (int i = 0; i < NUM_BIN; i++) {
        h_b[i] = 0;
    }

    // declare GPU memory pointers
    int * d_a;
    int * d_b;

    // allocate GPU memory
    hipMalloc((void **)&d_a, SIZE * sizeof(int));
    hipMalloc((void **)&d_b, NUM_BIN * sizeof(int));

    // transfer the arrays to the GPU
    hipMemcpy(d_a, h_a, SIZE * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, NUM_BIN * sizeof(int), hipMemcpyHostToDevice);

    // launch the kernel
    histogram_shared_memory <<<SIZE / 256, 256 >>>(d_b, d_a);

    // copy back the result from GPU
    hipMemcpy(h_b, d_b, NUM_BIN * sizeof(int), hipMemcpyDeviceToHost);
    printf("Histogram using 16 bin is: ");
        for (int i = 0; i < NUM_BIN; i++) {
            printf("bin %d: count %d\n", i, h_b[i]);
        }

    // free GPU memory allocation
    hipFree(d_a);
    hipFree(d_b);

    return 0;
}
