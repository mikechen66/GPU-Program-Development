
#include <hip/hip_runtime.h>
#include <stdio.h>

#define NUM_THREADS 10000
#define SIZE  10
#define BLOCK_WIDTH 100

__global__ void gpu_increment_without_atomic(int *d_a)
{
    // Calculate thread id for current thread
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // each thread increments elements wrapping at SIZE variable
    tid = tid % SIZE;
    d_a[tid] += 1;
}

int main(int argc, char **argv)
{

    printf("%d total threads in %d blocks writing into %d array elements\n",
        NUM_THREADS, NUM_THREADS / BLOCK_WIDTH, SIZE);

    // declare and allocate host memory
    int h_a[SIZE];
    const int ARRAY_BYTES = SIZE * sizeof(int);

    // declare and allocate GPU memory
    int * d_a;
    hipMalloc((void **)&d_a, ARRAY_BYTES);
    //Initialize GPU memory to zero
    hipMemset((void *)d_a, 0, ARRAY_BYTES);

    gpu_increment_without_atomic <<<NUM_THREADS / BLOCK_WIDTH, BLOCK_WIDTH >>>(d_a);

    // copy back the array to host memory
    hipMemcpy(h_a, d_a, ARRAY_BYTES, hipMemcpyDeviceToHost);

    printf("Number of times a particular Array index has been incremented without atomic add is: \n");
    for (int i = 0; i < SIZE; i++)
    {
        printf("index: %d --> %d times\n ", i, h_a[i]);
    }

    hipFree(d_a);
    return 0;
}